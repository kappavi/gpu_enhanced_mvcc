#include <iostream>
#include <vector>
#include <unordered_map>
#include <mutex>
#include <climits>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>

#define MAX_OBJECTS 1000
#define MAX_VERSIONS 1000

struct MVCCVersion {
    int value;
    int begin_ts;
    int end_ts;
    bool is_committed;
};

__global__ void parallel_write_kernel(
    int* object_ids,
    int* values,
    int n,
    MVCCVersion* versions,
    int* version_counts,
    int timestamp
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int obj_id = object_ids[idx];
        int val = values[idx];
        int version_idx = atomicAdd(&version_counts[obj_id], 1);
        MVCCVersion new_version;
        new_version.value = val;
        new_version.begin_ts = timestamp;
        new_version.end_ts = INT_MAX;
        new_version.is_committed = false;
        versions[obj_id * MAX_VERSIONS + version_idx] = new_version;
    }
}

void checkCudaError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("CUDA error at %s:%d: %s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA(call) checkCudaError(call, __FILE__, __LINE__)

class MVCCStore {
private:
    std::unordered_map<int, std::vector<MVCCVersion>> versions;
    std::mutex mutex;
    int current_timestamp;
    MVCCVersion* d_versions;
    int* d_version_counts;
    
public:
    MVCCStore() : current_timestamp(0) {
        hipMalloc(&d_versions, sizeof(MVCCVersion) * MAX_OBJECTS * MAX_VERSIONS);
        hipMalloc(&d_version_counts, sizeof(int) * MAX_OBJECTS);
        hipMemset(d_version_counts, 0, sizeof(int) * MAX_OBJECTS);
    }
    
    ~MVCCStore() {
        hipFree(d_versions);
        hipFree(d_version_counts);
    }
    
    void batch_write_gpu(const std::vector<int>& object_ids, const std::vector<int>& values) {
        int n = object_ids.size();
        int* d_object_ids;
        int* d_values;
        
        CHECK_CUDA(hipMalloc(&d_object_ids, sizeof(int) * n));
        CHECK_CUDA(hipMalloc(&d_values, sizeof(int) * n));
        CHECK_CUDA(hipMemcpy(d_object_ids, object_ids.data(), sizeof(int) * n, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_values, values.data(), sizeof(int) * n, hipMemcpyHostToDevice));
        
        int block_size = 256;
        int num_blocks = (n + block_size - 1) / block_size;
        
        parallel_write_kernel<<<num_blocks, block_size>>>(
            d_object_ids, d_values, n, d_versions, d_version_counts, ++current_timestamp
        );
        
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipFree(d_object_ids));
        CHECK_CUDA(hipFree(d_values));
    }
    
    void write(int object_id, int value) {
        std::lock_guard<std::mutex> lock(mutex);
        int ts = ++current_timestamp;
        if (!versions[object_id].empty()) {
            versions[object_id].back().end_ts = ts;
            versions[object_id].back().is_committed = true;
        }
        MVCCVersion new_version;
        new_version.value = value;
        new_version.begin_ts = ts;
        new_version.end_ts = INT_MAX;
        new_version.is_committed = false;
        versions[object_id].push_back(new_version);
    }

    int read(int object_id) {
        std::lock_guard<std::mutex> lock(mutex);
        if (versions.find(object_id) == versions.end() || versions[object_id].empty()) { return -1; }
        for (auto it = versions[object_id].rbegin(); it != versions[object_id].rend(); ++it) { if (it->is_committed) { return it->value; } }
        return -1;
    }

    int read_at_ts(int object_id, int ts) {
        std::lock_guard<std::mutex> lock(mutex);
        if (versions.find(object_id) == versions.end()) return -1;
        for (const auto& version : versions[object_id]) { if (version.begin_ts <= ts && ts < version.end_ts && version.is_committed) { return version.value; }}
        return -1;
    }
    void commit() {
        std::lock_guard<std::mutex> lock(mutex);
        for (auto& [object_id, object_versions] : versions) { if (!object_versions.empty() && !object_versions.back().is_committed) { object_versions.back().is_committed = true; } }
    }
    void rollback() {
        std::lock_guard<std::mutex> lock(mutex);
        for (auto& [object_id, object_versions] : versions) { if (!object_versions.empty() && !object_versions.back().is_committed) { object_versions.pop_back(); } }
    }
};

class Benchmark {
public:
    static void run_cpu_test(MVCCStore& store, int num_operations) {
        auto start = std::chrono::high_resolution_clock::now();
        
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dis(0, 999);
        
        for (int i = 0; i < num_operations; i++) {
            store.write(dis(gen), i);
            if (i % 100 == 0) store.commit();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        std::cout << "CPU test with " << num_operations << " operations took " << duration.count() / 1000.0 << "ms" << std::endl;
    }
    
    static void run_gpu_test(MVCCStore& store, int num_operations) {
        std::vector<int> warmup_ids(100);
        std::vector<int> warmup_vals(100);
        for (int i = 0; i < 100; i++) {
            warmup_ids[i] = i;
            warmup_vals[i] = i;
        }
        store.batch_write_gpu(warmup_ids, warmup_vals);
        CHECK_CUDA(hipDeviceSynchronize());
        
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dis(0, 999);
        
        std::vector<int> object_ids(num_operations);
        std::vector<int> values(num_operations);
        
        for (int i = 0; i < num_operations; i++) {
            object_ids[i] = dis(gen);
            values[i] = i;
        }
        
        CHECK_CUDA(hipDeviceSynchronize());
        
        auto start = std::chrono::high_resolution_clock::now();
        
        store.batch_write_gpu(object_ids, values);
        store.commit();
        
        CHECK_CUDA(hipDeviceSynchronize());
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        std::cout << "GPU test with " << num_operations << " operations took " << duration.count() / 1000.0 << "ms" << std::endl;
    }
};

int main() {
    MVCCStore store;

    // Basic test
    std::cout << "Running basic test..." << std::endl;
    store.write(1, 100);
    store.commit();
    store.write(1, 200);
    store.commit();
    store.write(1, 300);
    std::cout << "Latest value (committed): " << store.read(1) << std::endl;

    // Performance benchmarks
    std::cout << "\nRunning benchmarks..." << std::endl;
    
    std::cout << "\nSmall dataset (1000 operations):" << std::endl;
    Benchmark::run_cpu_test(store, 1000);
    Benchmark::run_gpu_test(store, 1000);
    
    std::cout << "\nMedium dataset (10000 operations):" << std::endl;
    Benchmark::run_cpu_test(store, 10000);
    Benchmark::run_gpu_test(store, 10000);
    
    std::cout << "\nLarge dataset (100000 operations):" << std::endl;
    Benchmark::run_cpu_test(store, 100000);
    Benchmark::run_gpu_test(store, 100000);
    
    return 0;
} 